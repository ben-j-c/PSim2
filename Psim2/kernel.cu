#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <stdexcept>

#include "kernel.cuh"

#define USE_SHARED

#define cudaErrorCheck(...) { cudaErrorCheckingFunction((__VA_ARGS__), __FILE__, __LINE__); } 
#define blockSize ((int) 64)
#define blocksToLoad ((int) 2)

static Particle *plist;
static int numpart;
static Particle *device_plist;
static Vector3 *host_pos;

static inline void cudaErrorCheckingFunction(hipError_t error, const char* file, int line, bool abort = true) {
	if (error != hipSuccess) {
		fprintf(stderr, "Cuda error: %s %s %d\n", hipGetErrorString(error), file, line);
		//throw std::runtime_error("CUDA error");
	}
}

static __device__ void setColour(Particle& p, int index, Vector3* outColor) {
	float magVel = magVector3(&p.v);
	//outColor[idx] = {fminf(1.0, 0.5/minDist) ,0.1, fminf(1.0f, minDist/0.5)};
	float r = fmaxf(fminf(magVel * 7, 1.0f), 0.1f);
	float g = fmaxf(fminf(magVel * 7 - 0.5, 1.0f), 0.1f);
	float b = fmaxf(fminf(magVel * 7 - 0.75, 1.0f), 0.1f);
	outColor[index] = { r, g, b };
}

static __global__ void gpu_doStepWithShared(Vector3 *outPos, Vector3 *outColor, Particle * nplist, int numP, float k0, float G, float timeStep) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	Vector3 newPos, newV;

	__shared__ Particle plist[blockSize*blocksToLoad];
	Particle self = nplist[idx];

	Vector3 force = { 0.0f,0.0f,0.0f };

	/*
	e.g., Block size = 8, sizeof(nplist) = 4*8 = 32, blocksToLoad = 2
	Blocks:	0        1        2        3
			|--------|--------|--------|--------|

	Each block will have  2 blocks loaded into shared memory.
	Block 1 threads will load on the second iteration:
	T0: nplist[16] and nplist[24]
	T1: 17 25
	T2: 18 26
	T3: 19 27
	T4: 20 28
	T5: 21 29
	T6: 22 30
	T7: 23 31

	T2 loads from:
	nplist:
	0        1        2        3
	|--------|--------|--X-----|--Y-----|
						 |        |
						 V        V
	into plist:       |--Z----- --W-----|
	*/

	//blockDim.x*blockSize;
	for (int block = 0; block < numP; block += blockSize * blocksToLoad) { //For every section of blocks to load into plist
		for (int i = 0; i < blocksToLoad; i++) {
			int offset = threadIdx.x + i * blockSize;
			plist[offset] = nplist[block + offset];
		}
		__syncthreads();
		for (int i = 0; i < blockSize*blocksToLoad; i++) {
			Vector3 r;
			copyVector3(&r, &self.pos);
			subVector3(&r, &plist[i].pos);
			float magR = magVector3(&r) + 0.05;

			float scaleFactor = -plist[i].m / (magR*magR*magR);

			Vector3 newForce = { 0.0f, 0.0f ,0.0f };
			copyVector3(&newForce, &r);
			scaleVector3(&newForce, G*scaleFactor);

			addVector3(&force, &newForce);
		}
	}

	
	newV.x = self.v.x + force.x*timeStep / self.m;
	
	newV.y = self.v.y + force.y*timeStep / self.m;
	
	newV.z = self.v.z + force.z*timeStep / self.m;

	float magNewV = magVector3(&newV);
	if (magNewV > 25) {
		normVector3(&newV);
		newV.x *= 25;
		newV.y *= 25;
		newV.z *= 25;
	}

	
	newPos.x = self.pos.x + timeStep * (self.v.x + newV.x)*0.5;
	newPos.y = self.pos.y + timeStep * (self.v.y + newV.y)*0.5;
	newPos.z = self.pos.z + timeStep * (self.v.z + newV.z)*0.5;

	if (self.isStationary) {
		copyVector3(&newPos, &self.pos);
		copyVector3(&newV, &self.v);
	}
	
	setColour(self, idx, outColor);
	copyVector3(&outPos[idx], &self.pos);
	copyVector3(&nplist[idx].pos, &newPos);
	copyVector3(&nplist[idx].v, &newV);
}

static __global__ void computeVelocity(Particle* parts) {
	int i = threadIdx.x + blockDim.x*blockIdx.x;

	float mag = magVector3(&parts[i].pos);
	Vector3 ortho = { -parts[i].pos.z, 0.0, parts[i].pos.x };
	normVector3(&ortho);
	scaleVector3(&ortho, mag / 75.0);
	copyVector3(&parts[i].v, &ortho);
}

static __global__ void loadOutputs(Vector3 * pos, Vector3 * colour, Particle * nplist) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	setColour(nplist[idx], idx, colour);
	copyVector3(&pos[idx], &nplist[idx].pos);
}

void DeviceFunctions::loadData(Vector3 * pos, Vector3 * colour) {
	dim3 blocks(numpart / blockSize, 1, 1);
	dim3 threadsPerBlock(blockSize, 1, 1);

	loadOutputs <<< blocks, threadsPerBlock >>> (pos, colour, device_plist);
	cudaErrorCheck(hipGetLastError());
	hipDeviceSynchronize();
}

Particle * DeviceFunctions::getPlist() {
	return plist;
}

Vector3* DeviceFunctions::getParticlePos(Vector3 *pos) {
	hipMemcpy(host_pos, pos, sizeof(Vector3)*numpart, hipMemcpyDeviceToHost);
	return host_pos;
}

void DeviceFunctions::doStep(float timestep, Vector3 *pos, Vector3 *colour) {
	dim3 blocks(numpart / blockSize, 1, 1);
	dim3 threadsPerBlock(blockSize, 1, 1);

	printf("Pos:    %#016llx\n", pos);
	printf("Colour: %#016llx\n", colour);
	printf("Plist:  %#016llx\n", device_plist);
	hipDeviceSynchronize();
	gpu_doStepWithShared <<< blocks, threadsPerBlock >>> (pos, colour, device_plist, numpart, 1.5, 0.0001, timestep);
	cudaErrorCheck(hipGetLastError());
	printf("--------------------------------\n");
}



static float randFloat() {
	return (float)rand() / RAND_MAX;
}

static float randRange(float a, float b) {
	return ((float)rand() / RAND_MAX)*(b - a) + a;
}

int DeviceFunctions::setup(int num) {
	numpart = ((num - 1) / (blockSize*blocksToLoad) + 1)*blockSize*blocksToLoad;
	static bool firstRun = true;
	if(firstRun)
		cudaErrorCheck(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
	printf("Startup...\n");
	printf("Given %d particles, rounding to %d - the nearest multiple of blocksize*blocksToLoad (%d * %d)\n", num, numpart, blockSize, blocksToLoad);
	printf("sizeof(Particle) = %d\n", sizeof(Particle));
	plist = (Particle*)malloc(sizeof(Particle)*numpart);
	host_pos = (Vector3*)malloc(sizeof(Vector3)*numpart);


	for (int i = 0; i < numpart; i++) {
		plist[i].pos = { randRange(-15,15),randRange(-5,5),randRange(-15,15) };
		plist[i].v = { 0.0, 0.0, 0.0 };
		plist[i].m = randFloat() / numpart;
		plist[i].q = randRange(0.001, 0.001) / numpart; //10% should be weakly negative. This should allow for clumping
		plist[i].isStationary = 0;
	}

	cudaErrorCheck(hipMalloc(&device_plist, sizeof(Particle)*numpart));
	cudaErrorCheck(hipMemcpy(device_plist, plist, sizeof(Particle)*numpart, hipMemcpyHostToDevice));

	dim3 blocks(numpart / blockSize, 1, 1);
	dim3 threadsPerBlock(blockSize, 1, 1);

	computeVelocity <<< blocks, threadsPerBlock >>> (device_plist);
	cudaErrorCheck(hipGetLastError());

	firstRun = false;
	return numpart;
}

#undef blockSize
int DeviceFunctions::shutdown() {
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	
	//cudaErrorCheck(hipFree(device_plist));
	device_plist = nullptr;
	free(plist);
	free(host_pos);
	return 0;
}
