#include "hip/hip_runtime.h"
#include "Vector3.cuh"
#include <hip/hip_runtime.h>

#ifdef __INTELLISENSE__
//#define __HIPCC__
#include <math_functions.h>
#endif // __INTELLISENSE__



Vector3* addVector3(Vector3* dst, Vector3* src) {
	dst->x = dst->x + src->x;
	dst->y = dst->y + src->y;
	dst->z = dst->z + src->z;
	return dst;
}

Vector3* scaleVector3(Vector3* dst, float src) {
	dst->x = dst->x * src;
	dst->y = dst->y * src;
	dst->z = dst->z * src;
	return dst;
}

Vector3* subVector3(Vector3* dst, Vector3* src) {
	dst->x = dst->x + src->x;
	dst->y = dst->y + src->y;
	dst->z = dst->z + src->z;
	return dst;
}

Vector3* normVector3(Vector3* dst) {
	float invSqrt = rsqrtf(dst->x*dst->x + dst->y*dst->y + dst->z*dst->z);
	return scaleVector3(dst, invSqrt);
}

float magVector3(Vector3* dst) {
	return sqrtf(dst->x*dst->x + dst->y*dst->y + dst->z*dst->z);
}

Vector3* copyVector3(Vector3* dst, Vector3* src) {
	dst->x = src->x;
	dst->y = src->y;
	dst->z = src->z;
	return dst;
}

Vector3* copyVector3(Vector3* dst, std::array<float, 3>& src) {
	dst->x = src[0];
	dst->y = src[1];
	dst->z = src[2];
	return dst;
}